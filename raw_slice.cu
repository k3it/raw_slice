#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <netdb.h>
#include <sys/socket.h>
#include <arpa/inet.h>
#include "gputimer.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


/* FILTER PARAMETERS

        Low pass Direct-form FIR
        Order 262144
        Window Kaiser
        Fs = 61440000 Hz
        Fpass = 190497.31 Hz
        Fstop = 192000.0 Hz
        Apass = 0.1 db
        Astop = 100 db

*/

// comment out to speed up execution by 30% 
//#define USE_DBL_PRECISION_FFT

#define DFT_BLOCK_SIZE 1048576    // 1M real samples
#define L_SIZE 786432             // Number of new input samples consumed per data block 

#define COMPLEX_SIGNAL_SIZE (DFT_BLOCK_SIZE/2 + 1)   // R2C FFT length

#define P_SIZE 262145             // FIR Length
#define V_SIZE 4                  // Overlap factor  V = N/(P-1)
#define SAMPLING_RATE 61440000    // Hz
#define D_size 128                // decimation factor
#define LO 14040078.125           // LO
//#define N_ROT  240300       // LO = 14080078.125 Hz  Nrot = round(COMPLEX_SIGNAL_SIZE*LO/V_SIZE*SAMPLING_RATE) * V_SIZE
#define SAMPLE_LEN 16       // 16 bit real samples

// mix to baseband formula
// new_index = (index >= rot) ? index - N_ROT : FFT_SIZE - N_ROT + index

// Complex multiplication
static __device__ inline hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b)
{
    hipfftComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

// Complex scale
static __device__ inline hipfftComplex ComplexScale(hipfftComplex a, float s)
{
    hipfftComplex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

static __device__ inline hipfftComplex ComplexAdd(hipfftComplex a, hipfftComplex b)
{
    hipfftComplex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}

__global__
void gpu_process_buffer(hipfftReal * d_signal, const short * d_buffer, hipfftReal * d_delay_line)
{
    
    //  This is the overlap-save routine and conversion from short to float
    //  Not sure about scaling factor 256.0 

    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // copy the last P-1 samples to the beginning ofthe window
    if (tid < P_SIZE - 1) {
        d_signal[tid] = d_delay_line[tid];

        // store the tail of the buffer in the delay line
        d_delay_line[tid] = d_buffer[L_SIZE - (P_SIZE -1) + tid ]/256.0f;
    } 
    else
    {
        // add buffer to the fft window
        d_signal[tid] = d_buffer[tid - (P_SIZE - 1)]/256.0f;
    }


}

__global__
void gpu_mix_and_convolve(const hipfftComplex *d_fft, const hipfftComplex *d_fir_fft, 
                                hipfftComplex * d_receiver, hipfftComplex * d_receiver2,
                                const int nrot, const float scale)
{
    const size_t numThreads = blockDim.x * gridDim.x;
    const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    // mix to baseband

    size_t new_index;
    //size_t new_index = (tid >= nrot) ? tid - nrot : COMPLEX_SIGNAL_SIZE - nrot + tid;
    //d_receiver[tid] = d_fft[new_index];

    for (int i = tid; i < COMPLEX_SIGNAL_SIZE; i += numThreads)
    {
        new_index = (i >= nrot) ? i - nrot : COMPLEX_SIGNAL_SIZE - nrot + i;
        d_receiver[i]  = d_fft[new_index];
        d_receiver[i] = ComplexScale(ComplexMul(d_receiver[i], d_fir_fft[i]), scale);
        
    }

}

__global__
void gpu_decimate(const hipfftComplex * d_receiver, hipfftComplex * d_slice)
{
    //const int numThreads = blockDim.x * gridDim.x;
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid > COMPLEX_SIGNAL_SIZE/D_size) return;

    hipfftComplex decimated_bin;
    decimated_bin.x = 0;
    decimated_bin.y = 0;

    // fold FFT back on itself
    for (int i = tid; i < COMPLEX_SIGNAL_SIZE; i += COMPLEX_SIGNAL_SIZE/D_size) 
    {
        decimated_bin = ComplexAdd(decimated_bin, d_receiver[i]);
    }

    d_slice[tid] = decimated_bin;
    
}

int
main(int argc, char **argv)
{

   
    // create buffers
    size_t fft_result_size = sizeof(hipfftComplex)*COMPLEX_SIGNAL_SIZE;
    size_t buffer_size = sizeof(short)*L_SIZE;
    size_t delay_line_size = sizeof(hipfftReal) * (P_SIZE - 1);
    size_t d_signal_size = sizeof(hipfftReal)*DFT_BLOCK_SIZE;

    size_t d_fir_size = sizeof(hipfftReal)*DFT_BLOCK_SIZE;
    size_t d_fir_fft_size = sizeof(hipfftComplex)*COMPLEX_SIGNAL_SIZE;

    size_t rx_td_size = sizeof(hipfftReal)*DFT_BLOCK_SIZE/D_size;


    // findCudaDevice(argc, (const char **)argv);
    // Set flag to enable zero copy access
    hipSetDeviceFlags(hipDeviceMapHost);

    GpuTimer timer;
    timer.Start();


    // Allocate host-device mapped memory for the fir and buffer
    
    unsigned char *h_buffer = NULL;  // input stream buffer for ADC samples
    hipfftReal *h_fir = NULL;   // host buffer for the FIR filter coefs
    hipfftReal *h_rx_td = NULL; // decimated and filtered signal goes here

    hipHostAlloc((void **)&h_fir, d_fir_size, hipHostMallocMapped); 
    hipHostAlloc((void **)&h_buffer, buffer_size, hipHostMallocMapped); 
    hipHostAlloc((void **)&h_rx_td, rx_td_size, hipHostMallocMapped); 
    
    hipfftReal *d_signal;  // time domain input signal for overlap-save
    hipfftComplex *d_fft;  // DFT of the input signal

    hipfftReal *d_delay_line;  // tail of each DFT window for overlap-save
    short *d_buffer; // device pointer to h_buffer
    hipfftReal *d_fir;   // device pointer to h_fir
    hipfftComplex *d_fir_fft;  // FFT of the FIR filter for fast convolution
    hipfftReal *d_rx_td;  // device pointer to h_rx_td


    // get device pointers for the mapped buffers
    //hipHostGetDevicePointer((void **)&d_signal, (void *) h_signal, 0);
    hipHostGetDevicePointer((void **)&d_fir, (void *) h_fir, 0);
    hipHostGetDevicePointer((void **)&d_buffer, (void *) h_buffer, 0);
    hipHostGetDevicePointer((void **)&d_rx_td, (void *) h_rx_td, 0);

    // allocate device memory for overlap-save
    hipMalloc((void **)&d_delay_line, delay_line_size);
    hipMalloc((void **)&d_signal, d_signal_size);
    hipMalloc((void **)&d_fft, fft_result_size);
    hipMalloc((void **)&d_fir_fft, d_fir_fft_size);

  
    // zero out buffers using the GPU
    // we can speed this up by using cudeMemsetAsync 
    hipMemset(d_signal, 0, d_signal_size);
    hipMemset(d_buffer, 0, buffer_size);
    hipMemset(d_delay_line, 0, delay_line_size);
    hipMemset(d_fir, 0, d_fir_size);
    hipMemset(d_rx_td, 0, rx_td_size);


    //allocate receiver
    hipfftComplex *d_receiver = NULL;
    hipfftComplex *d_slice = NULL;
    hipMalloc((void **)&d_receiver, fft_result_size);
    hipMalloc((void **)&d_slice, fft_result_size/D_size);

    hipfftComplex *d_receiver2 = NULL;
    hipMalloc((void **)&d_receiver2, fft_result_size);

    // ready to start processing input

    FILE *firfile;
    size_t samples_read=0;
    size_t total_samples_read=0;

    /* load filter coeffs */
    // fir file format one ASCII coeff per line
    char line[80];
    firfile=fopen("240khz.fir","r");
    //firfile=fopen("240khz-fir-float","r");
    //firfile=fopen("192khz-fir.fcf","r");

    

    /* did it open? */
    if (firfile == NULL)
    {
      fprintf(stderr, "ERROR opening filter file. aborting.\n");
      exit(1);
    }

    for (int i=0; i < P_SIZE; i++)
    {
        if (fgets(line, sizeof line, firfile) != NULL)
        {
            h_fir[i] = strtof(line, NULL);
            //printf("read coef: %.30f\n", h_fir[i]);
        }
        else
        {
          fprintf(stderr, "ERROR reading filter coefficients. aborting.\n");
          exit(1);
        }


    }

    // CUFFT plan
    //hipfftHandle planZ;  //double precision plan
    //hipfftPlan1d(&planZ, DFT_BLOCK_SIZE, HIPFFT_D2Z, 1);


     #ifdef USE_DBL_PRECISION_FFT
            hipfftHandle planZ2D; // double precision IFFT plan
            hipfftPlan1d(&planZ2D, DFT_BLOCK_SIZE, HIPFFT_Z2D, 1);
            
     #else
            hipfftHandle planC;  //single precition plan
            hipfftHandle planC2R; // single precision IFFT plan
            hipfftPlan1d(&planC, DFT_BLOCK_SIZE, HIPFFT_R2C, 1);
            hipfftPlan1d(&planC2R, DFT_BLOCK_SIZE/D_size, HIPFFT_C2R, 1);
            //hipfftDestroy(planZ);
     #endif

    //calclulate and store FFT of the FIR filter
    hipfftExecR2C(planC, d_fir, d_fir_fft);


    //hipDeviceSynchronize(); getLastCudaError("Kernel execution failed [ FIR FFT ]");

    // calculate FFT bin rotation value for the mixer 
    double nrot = round((double)LO*(COMPLEX_SIGNAL_SIZE-1) / ((double)V_SIZE*SAMPLING_RATE)) * V_SIZE;
    fprintf(stderr, "FFT rotation %d bins resulting in LO %.11g Hz\n", (int)nrot, nrot*SAMPLING_RATE/COMPLEX_SIGNAL_SIZE);

    int discard_size = (P_SIZE-1)/D_size;
    int td_size = L_SIZE/D_size;
    fprintf(stderr, "IFFT discard samples %d, keep %d, output sample size %d bytes(floating pt)\n", discard_size, td_size, sizeof(hipfftReal));

    timer.Stop();
    fprintf(stderr, "Setup complete in %g ms\n", timer.Elapsed());
    
    timer.Start();

    int skip = 2;  // number of initial frames to skip

    for(;;)
    {
        /* read from stdin until it's end */
        samples_read = fread(h_buffer, sizeof(short), L_SIZE, stdin);
        total_samples_read += samples_read;

        if (samples_read < buffer_size) 
        {
                if (feof(stdin)) break;
        }
        
        //fprintf(stdout, "read %d samples\n", samples_read);
        gpu_process_buffer<<<DFT_BLOCK_SIZE/1024, 1024>>>(d_signal, d_buffer, d_delay_line);
    
        // Check if kernel execution generated and error - this slows down execution
        // hipDeviceSynchronize(); getLastCudaError("Kernel execution failed [ gpu_process_buffer ]");

        #ifdef USE_DBL_PRECISION_FFT
                //call double precision real-to-complex FFT
                hipfftExecD2Z(planZ, d_signal, d_fft);
        #else 
                //call single precision real-to-complex FFT
                hipfftExecR2C(planC, (hipfftReal *) d_signal, (hipfftComplex *) d_fft);
        #endif

        gpu_mix_and_convolve<<<128, 1024>>>(d_fft, d_fir_fft, d_receiver, d_receiver2, (int)nrot, 1.0f/COMPLEX_SIGNAL_SIZE);

        gpu_decimate<<<COMPLEX_SIGNAL_SIZE/(1024*D_size), 1024>>>(d_receiver,d_slice);

        #ifdef USE_DBL_PRECISION_FFT
                hipfftExecZ2D(planZ2D, d_slice, d_rx_td);
        #else
                hipfftExecC2R(planC2R, (hipfftComplex *) d_slice, (hipfftReal *) d_rx_td);
        #endif

        //hipDeviceSynchronize(); getLastCudaError("Kernel execution failed [ IFFT ]");

        if (skip == 0) 
        {
            fwrite(h_rx_td+discard_size, sizeof(hipfftReal), td_size, stdout);
        }
        else
        {
            skip--;
        }
        
        
    }

    timer.Stop();
    float secs = (float)total_samples_read / SAMPLING_RATE;
    fprintf(stderr, "Processed %d samples (%g sec) signal in %g ms, performance ratio: %g\n", total_samples_read, secs, timer.Elapsed(), secs*1000/timer.Elapsed());

    #ifdef USE_DBL_PRECISION_FFT
            hipfftDestroy(planZ);
            hipfftDestroy(planZ2D);
    #else
            hipfftDestroy(planC);
            hipfftDestroy(planC2R);
    #endif

    hipFree(d_fft);
    hipFree(d_signal);
    hipFree(d_slice);
    hipFree(d_fir);
    hipFree(d_fir_fft);
    hipFree(d_buffer);
    hipFree(d_delay_line);
}